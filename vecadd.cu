#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void vecAddKernel(float *A,float * B,float *C,int n){
	int i=threadIdx.x+blockDim.x+blockIdx.x;
	if(i<n){
		C[i]=A[i]+B[i];
	}
}
// & address * value;
void vecAdd(float *h_A, float *h_B, float *h_C,int n){
	int size= n * sizeof(float);
	float *d_A,*d_B,*d_C;
	hipMalloc((void**)&d_A,size);
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&d_B,size);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
	hipError_t err=hipMalloc((void**)&d_C,size);
	/*if(err!=cudaSuccess){
		printf("error");
		exit(EXIT_FAILURE);
	}*/

	vecAddKernel<<<ceil(n/256.0),256>>>(d_A,d_B,d_C,n);
	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
	/*for(i=0;i<n;i++){
		h_C[i]=h_A[i]+h_B[i];
		printf("%lf",h_C[i]);
	}*/
	hipFree(d_C);
	hipFree(d_A);
	hipFree(d_B);
}
int main(){
	float *h_A,*h_B,*h_C;
	int n=4;

	h_A = (float *)malloc(n* sizeof(float));
	h_B = (float *)malloc(n* sizeof(float));
	h_C = (float *)malloc(n* sizeof(float));
	vecAdd(h_A,h_B,h_C,n);
	for(int i=0;i<n;i++){
		
		printf("%f ",h_C[i]);
	}
	free(h_A);
	free(h_B);
	free(h_C);
}
